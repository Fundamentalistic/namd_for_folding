#include "hip/hip_runtime.h"

#include "ComputeNonbondedCUDAKernel.h"
#include <stdio.h>

#ifdef NAMD_CUDA


__constant__ unsigned int const_exclusions[MAX_CONST_EXCLUSIONS];

static unsigned int *overflow_exclusions;

#define SET_EXCL(EXCL,BASE,DIFF) \
         (EXCL)[((BASE)+(DIFF))>>5] |= (1<<(((BASE)+(DIFF))&31))

void cuda_bind_exclusions(const unsigned int *t, int n) {

  hipMalloc((void**) &overflow_exclusions, n*sizeof(unsigned int));
  cuda_errcheck("malloc overflow_exclusions");
  hipMemcpy(overflow_exclusions, t,
		n*sizeof(unsigned int), hipMemcpyHostToDevice);
  cuda_errcheck("memcpy to overflow_exclusions");
  int nconst = ( n < MAX_CONST_EXCLUSIONS ? n : MAX_CONST_EXCLUSIONS );
  hipMemcpyToSymbol(HIP_SYMBOL(const_exclusions), t, nconst*sizeof(unsigned int), 0);
  cuda_errcheck("memcpy to const_exclusions");
}


texture<float2, 1, hipReadModeElementType> lj_table;
int lj_table_size;

void cuda_bind_lj_table(const float2 *t, int _lj_table_size) {
    static float2 *ct;
    static int lj_table_alloc;
    lj_table_size = _lj_table_size;
    if ( ct && lj_table_alloc < lj_table_size ) {
      hipFree(ct);
      cuda_errcheck("freeing lj table");
      ct = 0;
    }
    if ( ! ct ) {
      lj_table_alloc = lj_table_size;
      hipMalloc((void**) &ct, lj_table_size*lj_table_size*sizeof(float2));
      cuda_errcheck("allocating lj table");
    }
    hipMemcpy(ct, t, lj_table_size*lj_table_size*sizeof(float2),
                                            hipMemcpyHostToDevice);
    cuda_errcheck("memcpy to lj table");

    lj_table.normalized = false;
    lj_table.addressMode[0] = hipAddressModeClamp;
    lj_table.filterMode = hipFilterModePoint;

    hipBindTexture((size_t*)0, lj_table, ct,
        lj_table_size*lj_table_size*sizeof(float2));
    cuda_errcheck("binding lj table to texture");
}


texture<float4, 1, hipReadModeElementType> force_table;
texture<float4, 1, hipReadModeElementType> energy_table;

void cuda_bind_force_table(const float4 *t, const float4 *et) {
    static hipArray *ct;
    static hipArray *ect;
    if ( ! ct ) {
      hipMallocArray(&ct, &force_table.channelDesc, FORCE_TABLE_SIZE, 1);
      cuda_errcheck("allocating force table");
    }
    if ( ! ect ) {
      hipMallocArray(&ect, &energy_table.channelDesc, FORCE_TABLE_SIZE, 1);
      cuda_errcheck("allocating energy table");
    }
    hipMemcpyToArray(ct, 0, 0, t, FORCE_TABLE_SIZE*sizeof(float4), hipMemcpyHostToDevice);
    // hipMemcpy(ct, t, FORCE_TABLE_SIZE*sizeof(float4), hipMemcpyHostToDevice);
    cuda_errcheck("memcpy to force table");
    hipMemcpyToArray(ect, 0, 0, et, FORCE_TABLE_SIZE*sizeof(float4), hipMemcpyHostToDevice);
    cuda_errcheck("memcpy to energy table");

    force_table.normalized = true;
    force_table.addressMode[0] = hipAddressModeClamp;
    force_table.addressMode[1] = hipAddressModeClamp;
    force_table.filterMode = hipFilterModeLinear;

    energy_table.normalized = true;
    energy_table.addressMode[0] = hipAddressModeClamp;
    energy_table.addressMode[1] = hipAddressModeClamp;
    energy_table.filterMode = hipFilterModeLinear;

    hipBindTextureToArray(force_table, ct);
    cuda_errcheck("binding force table to texture");

    hipBindTextureToArray(energy_table, ect);
    cuda_errcheck("binding energy table to texture");
}

static int patch_pairs_size;
static patch_pair *patch_pairs;
static float *virial_buffers;  // one per patch pair
static float *slow_virial_buffers;  // one per patch pair

static int block_flags_size;
static unsigned int *block_flags;

static int force_lists_size;
static force_list *force_lists;
static unsigned int *force_list_counters;

static int force_buffers_size;
static float4 *force_buffers;
static float4 *slow_force_buffers;

static int atoms_size;
static atom *atoms;
static atom_param *atom_params;
static float4 *forces;
static float4 *slow_forces;
static float *virials;  // one per patch
static float *slow_virials;  // one per patch

static int patch_pairs_alloc;
static int block_flags_alloc;
static int force_buffers_alloc;
static int force_lists_alloc;
static int atoms_alloc;

static int max_atoms_per_patch;

// static hipStream_t stream;
hipStream_t stream;
 
void cuda_init() {
  forces = 0;
  slow_forces = 0;
  virials = 0;
  slow_virials = 0;
  atom_params = 0;
  atoms = 0;
  force_buffers = 0;
  slow_force_buffers = 0;
  force_lists = 0;
  force_list_counters = 0;
  patch_pairs = 0;
  virial_buffers = 0;
  slow_virial_buffers = 0;
  block_flags = 0;

  patch_pairs_alloc = 0;
  block_flags_alloc = 0;
  force_buffers_alloc = 0;
  force_lists_alloc = 0;
  atoms_alloc = 0;

  hipStreamCreate(&stream);
  cuda_errcheck("hipStreamCreate");
}

void cuda_bind_patch_pairs(const patch_pair *pp, int npp,
                        const force_list *fl, int nfl,
                        int atoms_size_p, int force_buffers_size_p,
                        int block_flags_size_p, int max_atoms_per_patch_p) {

  patch_pairs_size = npp;
  force_buffers_size = force_buffers_size_p;
  force_lists_size = nfl;
  atoms_size = atoms_size_p;
  block_flags_size = block_flags_size_p;
  max_atoms_per_patch = max_atoms_per_patch_p;

#if 0
 printf("%d %d %d %d %d %d %d %d\n",
      patch_pairs_size , patch_pairs_alloc ,
      force_buffers_size , force_buffers_alloc ,
      force_lists_size , force_lists_alloc ,
      atoms_size , atoms_alloc );
#endif

 if ( patch_pairs_size > patch_pairs_alloc ||
      block_flags_size > block_flags_alloc ||
      force_buffers_size > force_buffers_alloc ||
      force_lists_size > force_lists_alloc ||
      atoms_size > atoms_alloc ) {

  block_flags_alloc = (int) (1.2 * block_flags_size);
  patch_pairs_alloc = (int) (1.2 * patch_pairs_size);
  force_buffers_alloc = (int) (1.2 * force_buffers_size);
  force_lists_alloc = (int) (1.2 * force_lists_size);
  atoms_alloc = (int) (1.2 * atoms_size);

  // if ( forces ) hipFree(forces);
  // if ( slow_forces ) hipFree(slow_forces);
  forces = slow_forces = 0;
  if ( atom_params ) hipFree(atom_params);
  if ( atoms ) hipFree(atoms);
  if ( force_buffers ) hipFree(force_buffers);
  if ( slow_force_buffers ) hipFree(slow_force_buffers);
  if ( force_lists ) hipFree(force_lists);
  if ( force_list_counters ) hipFree(force_list_counters);
  // if ( virials ) hipFree(virials);
  virials = slow_virials = 0;
  if ( patch_pairs ) hipFree(patch_pairs);
  if ( virial_buffers ) hipFree(virial_buffers);
  if ( slow_virial_buffers ) hipFree(slow_virial_buffers);
  if ( block_flags ) hipFree(block_flags);
  cuda_errcheck("free everything");

#if 0
  int totalmem = patch_pairs_alloc * sizeof(patch_pair) +
		force_lists_alloc * sizeof(force_list) +
		2 * force_buffers_alloc * sizeof(float4) +
		atoms_alloc * sizeof(atom) +
		atoms_alloc * sizeof(atom_param) +
		2 * atoms_alloc * sizeof(float4);
  // printf("allocating %d MB of memory on GPU\n", totalmem >> 20);
  printf("allocating %d MB of memory for block flags\n",
				(block_flags_alloc * 4) >> 20);
#endif

  hipMalloc((void**) &block_flags, block_flags_alloc * 4);
  hipMalloc((void**) &virial_buffers, patch_pairs_alloc * 16*sizeof(float));
  hipMalloc((void**) &slow_virial_buffers, patch_pairs_alloc * 16*sizeof(float));
  hipMalloc((void**) &patch_pairs, patch_pairs_alloc * sizeof(patch_pair));
  // hipMalloc((void**) &virials, 2 * force_lists_alloc * 16*sizeof(float));
  // slow_virials = virials + force_lists_size * 16;
  hipMalloc((void**) &force_lists, force_lists_alloc * sizeof(force_list));
  hipMalloc((void**) &force_list_counters, force_lists_alloc * sizeof(unsigned int));
  hipMalloc((void**) &force_buffers, force_buffers_alloc * sizeof(float4));
  hipMalloc((void**) &slow_force_buffers, force_buffers_alloc * sizeof(float4));
  hipMalloc((void**) &atoms, atoms_alloc * sizeof(atom));
  hipMalloc((void**) &atom_params, atoms_alloc * sizeof(atom_param));
  // hipMalloc((void**) &forces, atoms_alloc * sizeof(float4));
  // hipMalloc((void**) &slow_forces, atoms_alloc * sizeof(float4));
  cuda_errcheck("malloc everything");

 }

  hipMemcpy(patch_pairs, pp, npp * sizeof(patch_pair),
				hipMemcpyHostToDevice);
  cuda_errcheck("memcpy to patch_pairs");

  hipMemcpy(force_lists, fl, nfl * sizeof(force_list),
				hipMemcpyHostToDevice);
  cuda_errcheck("memcpy to force_lists");

  hipMemset(force_list_counters, 0, nfl * sizeof(unsigned int));
  cuda_errcheck("memset force_list_counters");
}

void cuda_bind_atom_params(const atom_param *t) {
  hipMemcpyAsync(atom_params, t, atoms_size * sizeof(atom_param),
				hipMemcpyHostToDevice, stream);
  cuda_errcheck("memcpy to atom_params");
}

void cuda_bind_atoms(const atom *a) {
  cuda_errcheck("before memcpy to atoms");
  hipMemcpyAsync(atoms, a, atoms_size * sizeof(atom),
				hipMemcpyHostToDevice, stream);
  cuda_errcheck("memcpy to atoms");
}

void cuda_bind_forces(float4 *f, float4 *f_slow) {
  hipHostGetDevicePointer(&forces, f, 0);
  cuda_errcheck("hipHostGetDevicePointer forces");
  hipHostGetDevicePointer(&slow_forces, f_slow, 0);
  cuda_errcheck("hipHostGetDevicePointer slow_forces");
}

void cuda_bind_virials(float *v) {
  hipHostGetDevicePointer(&virials, v, 0);
  cuda_errcheck("hipHostGetDevicePointer virials");
  slow_virials = virials + force_lists_size*16;
}

#if 0
void cuda_load_forces(float4 *f, float4 *f_slow, int begin, int count) {
  // printf("load forces %d %d %d\n",begin,count,atoms_size);
  hipMemcpyAsync(f+begin, forces+begin, count * sizeof(float4),
				hipMemcpyDeviceToHost, stream);
  if ( f_slow ) {
    hipMemcpyAsync(f_slow+begin, slow_forces+begin, count * sizeof(float4),
				hipMemcpyDeviceToHost, stream);
  }
  cuda_errcheck("memcpy from forces");
}

void cuda_load_virials(float *v, int doSlow) {
  int count = force_lists_size;
  if ( doSlow ) count *= 2;
  hipMemcpyAsync(v, virials, count * 16*sizeof(float),
				hipMemcpyDeviceToHost, stream);
  cuda_errcheck("memcpy from virials");
}
#endif

#if 0
__host__ __device__ static int3 patch_coords_from_id(
        dim3 PATCH_GRID, int id) {

  return make_int3( id % PATCH_GRID.x,
                ( id / PATCH_GRID.x ) % PATCH_GRID.y,
                id / ( PATCH_GRID.x * PATCH_GRID.y ) );
}

__host__ __device__ static int patch_id_from_coords(
        dim3 PATCH_GRID, int3 coords) {

  // handles periodic boundaries
  int x = (coords.x + 4 * PATCH_GRID.x) % PATCH_GRID.x;
  int y = (coords.y + 4 * PATCH_GRID.y) % PATCH_GRID.y;
  int z = (coords.z + 4 * PATCH_GRID.z) % PATCH_GRID.z;

  return ( z * PATCH_GRID.y + y ) * PATCH_GRID.x + x;
}

__host__ __device__ static int3 patch_offset_from_neighbor(int neighbor) {

  // int3 coords = patch_coords_from_id(make_uint3(3,3,3), 13 + neighbor);
  int3 coords = patch_coords_from_id(make_uint3(3,3,3), neighbor);
  return make_int3(coords.x - 1, coords.y - 1, coords.z - 1);

}
#endif
 
#define BLOCK_SIZE 128
#define SHARED_SIZE 32


#define MAKE_PAIRLIST
#define DO_SLOW
#define DO_ENERGY
#include "ComputeNonbondedCUDAKernelBase.h"
#undef DO_ENERGY
#include "ComputeNonbondedCUDAKernelBase.h"
#undef DO_SLOW
#define DO_ENERGY
#include "ComputeNonbondedCUDAKernelBase.h"
#undef DO_ENERGY
#include "ComputeNonbondedCUDAKernelBase.h"
#undef MAKE_PAIRLIST
#define DO_SLOW
#define DO_ENERGY
#include "ComputeNonbondedCUDAKernelBase.h"
#undef DO_ENERGY
#include "ComputeNonbondedCUDAKernelBase.h"
#undef DO_SLOW
#define DO_ENERGY
#include "ComputeNonbondedCUDAKernelBase.h"
#undef DO_ENERGY
#include "ComputeNonbondedCUDAKernelBase.h"


void cuda_nonbonded_forces(float3 lata, float3 latb, float3 latc,
		float cutoff2, float plcutoff2,
		int cbegin, int ccount, int pbegin, int pcount,
		int doSlow, int doEnergy, int usePairlists, int savePairlists) {

 if ( ccount ) {
   if ( usePairlists ) {
     if ( ! savePairlists ) plcutoff2 = 0.;
   } else {
     plcutoff2 = cutoff2;
   }
   int grid_dim = 65535;  // maximum allowed
   for ( int cstart = 0; cstart < ccount; cstart += grid_dim ) {
     if ( grid_dim > ccount - cstart ) grid_dim = ccount - cstart;
     // printf("%d %d %d\n",cbegin+cstart,grid_dim,patch_pairs_size);

#define CALL(X) X<<< grid_dim, BLOCK_SIZE, 0, stream \
	>>>(patch_pairs+cbegin+cstart,atoms,atom_params,force_buffers, \
	     (doSlow?slow_force_buffers:0), block_flags, \
             virial_buffers, (doSlow?slow_virial_buffers:0), \
             overflow_exclusions, force_list_counters, force_lists, \
             forces, virials, \
             (doSlow?slow_forces:0), (doSlow?slow_virials:0), \
             lj_table_size, \
	     lata, latb, latc, cutoff2, plcutoff2, doSlow)

     if ( doEnergy ) {
       if ( doSlow ) {
         if ( plcutoff2 != 0. ) CALL(dev_nonbonded_slow_energy_pairlist);
         else CALL(dev_nonbonded_slow_energy);
       } else {
         if ( plcutoff2 != 0. ) CALL(dev_nonbonded_energy_pairlist);
         else CALL(dev_nonbonded_energy);
       }
     } else {
       if ( doSlow ) {
         if ( plcutoff2 != 0. ) CALL(dev_nonbonded_slow_pairlist);
         else CALL(dev_nonbonded_slow);
       } else {
         if ( plcutoff2 != 0. ) CALL(dev_nonbonded_pairlist);
         else CALL(dev_nonbonded);
       }
     }

     cuda_errcheck("dev_nonbonded");
   }
 }

#if 0
 if ( pcount ) {
  // printf("%d %d %d\n",pbegin,pcount,force_lists_size);
  dev_sum_forces<<< pcount, BLOCK_SIZE, 0, stream
	>>>(atoms,force_lists+pbegin,force_buffers,
                virial_buffers,forces,virials);
  if ( doSlow ) {
    dev_sum_forces<<< pcount, BLOCK_SIZE, 0, stream
	>>>(atoms,force_lists+pbegin,slow_force_buffers,
                slow_virial_buffers,slow_forces,slow_virials);
  }
  cuda_errcheck("dev_sum_forces");
 }
#endif

}


int cuda_stream_finished() {
  return ( hipStreamQuery(stream) == hipSuccess );
}


#else  // NAMD_CUDA

// for make depends
#include "ComputeNonbondedCUDAKernelBase.h"

#endif  // NAMD_CUDA

