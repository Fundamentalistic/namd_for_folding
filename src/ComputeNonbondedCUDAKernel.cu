#include "hip/hip_runtime.h"

#include "ComputeNonbondedCUDAKernel.h"
#include <stdio.h>

#ifdef NAMD_CUDA


__constant__ unsigned int exclusions[MAX_EXCLUSIONS];

#define SET_EXCL(EXCL,BASE,DIFF) \
         (EXCL)[((BASE)+(DIFF))>>5] |= (1<<(((BASE)+(DIFF))&31))

void cuda_bind_exclusions(const unsigned int *t, int n) {

  hipMemcpyToSymbol(HIP_SYMBOL(exclusions), t, n*sizeof(unsigned int), 0);
  cuda_errcheck("memcpy to exclusions");
}


texture<float4, 1, hipReadModeElementType> force_table;

void cuda_bind_force_table(const float4 *t) {
    static hipArray *ct;
    if ( ! ct ) {
      hipMallocArray(&ct, &force_table.channelDesc, FORCE_TABLE_SIZE, 1);
      cuda_errcheck("allocating force table");
    }
    hipMemcpyToArray(ct, 0, 0, t, FORCE_TABLE_SIZE*sizeof(float4), hipMemcpyHostToDevice);
    // hipMemcpy(ct, t, FORCE_TABLE_SIZE*sizeof(float4), hipMemcpyHostToDevice);
    cuda_errcheck("memcpy to force table");

    force_table.normalized = true;
    force_table.addressMode[0] = hipAddressModeClamp;
    force_table.addressMode[1] = hipAddressModeClamp;
    force_table.filterMode = hipFilterModeLinear;

    hipBindTextureToArray(force_table, ct);
    cuda_errcheck("binding force table to texture");
}

static int patch_pairs_size;
static patch_pair *patch_pairs;
static float *virial_buffers;  // one per patch pair
static float *slow_virial_buffers;  // one per patch pair

static int block_flags_size;
static unsigned int *block_flags;

static int force_lists_size;
static force_list *force_lists;
static unsigned int *force_list_counters;

static int force_buffers_size;
static float4 *force_buffers;
static float4 *slow_force_buffers;

static int atoms_size;
static atom *atoms;
static atom_param *atom_params;
static float4 *forces;
static float4 *slow_forces;
static float *virials;  // one per patch
static float *slow_virials;  // one per patch

static int patch_pairs_alloc;
static int block_flags_alloc;
static int force_buffers_alloc;
static int force_lists_alloc;
static int atoms_alloc;

static int max_atoms_per_patch;

// static hipStream_t stream;
hipStream_t stream;
 
void cuda_init() {
  forces = 0;
  slow_forces = 0;
  virials = 0;
  slow_virials = 0;
  atom_params = 0;
  atoms = 0;
  force_buffers = 0;
  slow_force_buffers = 0;
  force_lists = 0;
  force_list_counters = 0;
  patch_pairs = 0;
  virial_buffers = 0;
  slow_virial_buffers = 0;
  block_flags = 0;

  patch_pairs_alloc = 0;
  block_flags_alloc = 0;
  force_buffers_alloc = 0;
  force_lists_alloc = 0;
  atoms_alloc = 0;

  hipStreamCreate(&stream);
  cuda_errcheck("hipStreamCreate");
}

void cuda_bind_patch_pairs(const patch_pair *pp, int npp,
                        const force_list *fl, int nfl,
                        int atoms_size_p, int force_buffers_size_p,
                        int block_flags_size_p, int max_atoms_per_patch_p) {

  patch_pairs_size = npp;
  force_buffers_size = force_buffers_size_p;
  force_lists_size = nfl;
  atoms_size = atoms_size_p;
  block_flags_size = block_flags_size_p;
  max_atoms_per_patch = max_atoms_per_patch_p;

#if 0
 printf("%d %d %d %d %d %d %d %d\n",
      patch_pairs_size , patch_pairs_alloc ,
      force_buffers_size , force_buffers_alloc ,
      force_lists_size , force_lists_alloc ,
      atoms_size , atoms_alloc );
#endif

 if ( patch_pairs_size > patch_pairs_alloc ||
      block_flags_size > block_flags_alloc ||
      force_buffers_size > force_buffers_alloc ||
      force_lists_size > force_lists_alloc ||
      atoms_size > atoms_alloc ) {

  block_flags_alloc = (int) (1.2 * block_flags_size);
  patch_pairs_alloc = (int) (1.2 * patch_pairs_size);
  force_buffers_alloc = (int) (1.2 * force_buffers_size);
  force_lists_alloc = (int) (1.2 * force_lists_size);
  atoms_alloc = (int) (1.2 * atoms_size);

  if ( forces ) hipFree(forces);
  if ( slow_forces ) hipFree(slow_forces);
  if ( atom_params ) hipFree(atom_params);
  if ( atoms ) hipFree(atoms);
  if ( force_buffers ) hipFree(force_buffers);
  if ( slow_force_buffers ) hipFree(slow_force_buffers);
  if ( force_lists ) hipFree(force_lists);
  if ( force_list_counters ) hipFree(force_list_counters);
  if ( virials ) hipFree(virials);
  if ( patch_pairs ) hipFree(patch_pairs);
  if ( virial_buffers ) hipFree(virial_buffers);
  if ( slow_virial_buffers ) hipFree(slow_virial_buffers);
  if ( block_flags ) hipFree(block_flags);
  cuda_errcheck("free everything");

#if 0
  int totalmem = patch_pairs_alloc * sizeof(patch_pair) +
		force_lists_alloc * sizeof(force_list) +
		2 * force_buffers_alloc * sizeof(float4) +
		atoms_alloc * sizeof(atom) +
		atoms_alloc * sizeof(atom_param) +
		2 * atoms_alloc * sizeof(float4);
  // printf("allocating %d MB of memory on GPU\n", totalmem >> 20);
  printf("allocating %d MB of memory for block flags\n",
				(block_flags_alloc * 4) >> 20);
#endif

  hipMalloc((void**) &block_flags, block_flags_alloc * 4);
  hipMalloc((void**) &virial_buffers, patch_pairs_alloc * 16*sizeof(float));
  hipMalloc((void**) &slow_virial_buffers, patch_pairs_alloc * 16*sizeof(float));
  hipMalloc((void**) &patch_pairs, patch_pairs_alloc * sizeof(patch_pair));
  hipMalloc((void**) &virials, 2 * force_lists_alloc * 16*sizeof(float));
  slow_virials = virials + force_lists_size * 16;
  hipMalloc((void**) &force_lists, force_lists_alloc * sizeof(force_list));
  hipMalloc((void**) &force_list_counters, force_lists_alloc * sizeof(unsigned int));
  hipMalloc((void**) &force_buffers, force_buffers_alloc * sizeof(float4));
  hipMalloc((void**) &slow_force_buffers, force_buffers_alloc * sizeof(float4));
  hipMalloc((void**) &atoms, atoms_alloc * sizeof(atom));
  hipMalloc((void**) &atom_params, atoms_alloc * sizeof(atom_param));
  hipMalloc((void**) &forces, atoms_alloc * sizeof(float4));
  hipMalloc((void**) &slow_forces, atoms_alloc * sizeof(float4));
  cuda_errcheck("malloc everything");

 }

  hipMemcpy(patch_pairs, pp, npp * sizeof(patch_pair),
				hipMemcpyHostToDevice);
  cuda_errcheck("memcpy to patch_pairs");

  hipMemcpy(force_lists, fl, nfl * sizeof(force_list),
				hipMemcpyHostToDevice);
  cuda_errcheck("memcpy to force_lists");

  hipMemset(force_list_counters, 0, nfl * sizeof(unsigned int));
  cuda_errcheck("memset force_list_counters");
}

void cuda_bind_atom_params(const atom_param *t) {
  hipMemcpyAsync(atom_params, t, atoms_size * sizeof(atom_param),
				hipMemcpyHostToDevice, stream);
  cuda_errcheck("memcpy to atom_params");
}

void cuda_bind_atoms(const atom *a) {
  cuda_errcheck("before memcpy to atoms");
  hipMemcpyAsync(atoms, a, atoms_size * sizeof(atom),
				hipMemcpyHostToDevice, stream);
  cuda_errcheck("memcpy to atoms");
}

void cuda_load_forces(float4 *f, float4 *f_slow, int begin, int count) {
  // printf("load forces %d %d %d\n",begin,count,atoms_size);
  hipMemcpyAsync(f+begin, forces+begin, count * sizeof(float4),
				hipMemcpyDeviceToHost, stream);
  if ( f_slow ) {
    hipMemcpyAsync(f_slow+begin, slow_forces+begin, count * sizeof(float4),
				hipMemcpyDeviceToHost, stream);
  }
  cuda_errcheck("memcpy from forces");
}

void cuda_load_virials(float *v, int doSlow) {
  int count = force_lists_size;
  if ( doSlow ) count *= 2;
  hipMemcpyAsync(v, virials, count * 16*sizeof(float),
				hipMemcpyDeviceToHost, stream);
  cuda_errcheck("memcpy from virials");
}

#if 0
__host__ __device__ static int3 patch_coords_from_id(
        dim3 PATCH_GRID, int id) {

  return make_int3( id % PATCH_GRID.x,
                ( id / PATCH_GRID.x ) % PATCH_GRID.y,
                id / ( PATCH_GRID.x * PATCH_GRID.y ) );
}

__host__ __device__ static int patch_id_from_coords(
        dim3 PATCH_GRID, int3 coords) {

  // handles periodic boundaries
  int x = (coords.x + 4 * PATCH_GRID.x) % PATCH_GRID.x;
  int y = (coords.y + 4 * PATCH_GRID.y) % PATCH_GRID.y;
  int z = (coords.z + 4 * PATCH_GRID.z) % PATCH_GRID.z;

  return ( z * PATCH_GRID.y + y ) * PATCH_GRID.x + x;
}

__host__ __device__ static int3 patch_offset_from_neighbor(int neighbor) {

  // int3 coords = patch_coords_from_id(make_uint3(3,3,3), 13 + neighbor);
  int3 coords = patch_coords_from_id(make_uint3(3,3,3), neighbor);
  return make_int3(coords.x - 1, coords.y - 1, coords.z - 1);

}
#endif
 
#define BLOCK_SIZE 128
#define SHARED_SIZE 32

__device__ __forceinline__ static void dev_sum_forces(
        const int force_list_index,
	const atom *atoms,
	const force_list *force_lists,
	const float4 *force_buffers,
	const float *virial_buffers,
	float4 *forces, float *virials);

__global__ static void dev_nonbonded(
	const patch_pair *patch_pairs,
	const atom *atoms,
	const atom_param *atom_params,
	float4 *force_buffers,
	float4 *slow_force_buffers,
	unsigned int *block_flags,
	float *virial_buffers,
	float *slow_virial_buffers,
        unsigned int *force_list_counters,
        const force_list *force_lists,
        float4 *forces, float *virials,
        float4 *slow_forces, float *slow_virials,
        float3 lata, float3 latb, float3 latc,
	float cutoff2, float plcutoff2, int doSlow) {
// call with one block per patch_pair
// call with BLOCK_SIZE threads per block
// call with no shared memory

#ifdef __DEVICE_EMULATION__
  #define myPatchPair (*(patch_pair*)(&pp.i))
#else
  #define myPatchPair pp.pp
#endif
  __shared__ union {
#ifndef __DEVICE_EMULATION__
    patch_pair pp;
#endif
    unsigned int i[PATCH_PAIR_SIZE];
  } pp;

 { // start of nonbonded calc

  #define pl plu.c
  __shared__ union {
    unsigned int i[BLOCK_SIZE];
    char c[4*BLOCK_SIZE];
  } plu;

  volatile __shared__ union {
    float a2d[32][3];
    float a1d[32*3];
  } sumf;

  volatile __shared__ union {
    float a2d[32][3];
    float a1d[32*3];
  } sumf_slow;

#ifdef __DEVICE_EMULATION__
  #define jpqs ((atom*)(jpqu.i))
#else
  #define jpqs jpqu.d
#endif
  __shared__ union {
#ifndef __DEVICE_EMULATION__
    atom d[SHARED_SIZE];
#endif
    unsigned int i[4*SHARED_SIZE];
    float f[4*SHARED_SIZE];
  } jpqu;

#ifdef __DEVICE_EMULATION__
  #define japs ((atom_param*)(japu.i))
#else
  #define japs japu.d
#endif
  __shared__ union {
#ifndef __DEVICE_EMULATION__
    atom_param d[SHARED_SIZE];
#endif
    unsigned int i[4*SHARED_SIZE];
  } japu;

  if ( threadIdx.x < PATCH_PAIR_USED ) {
    unsigned int tmp = ((unsigned int*)patch_pairs)[
			PATCH_PAIR_SIZE*blockIdx.x+threadIdx.x];
    pp.i[threadIdx.x] = tmp;
  }

  if ( threadIdx.x < 96 ) { // initialize net force in shared memory
    sumf.a1d[threadIdx.x] = 0.f;
    sumf_slow.a1d[threadIdx.x] = 0.f;
  }

  __syncthreads();

  // convert scaled offset with current lattice
  if ( threadIdx.x == 0 ) {
    float offx = myPatchPair.offset.x * lata.x
               + myPatchPair.offset.y * latb.x
               + myPatchPair.offset.z * latc.x;
    float offy = myPatchPair.offset.x * lata.y
               + myPatchPair.offset.y * latb.y
               + myPatchPair.offset.z * latc.y;
    float offz = myPatchPair.offset.x * lata.z
               + myPatchPair.offset.y * latb.z
               + myPatchPair.offset.z * latc.z;
    myPatchPair.offset.x = offx;
    myPatchPair.offset.y = offy;
    myPatchPair.offset.z = offz;
  }

  __syncthreads();

  for ( int blocki = 0;
        blocki < myPatchPair.patch1_force_size;
        blocki += BLOCK_SIZE ) {

  atom ipq;
  struct {
    float sqrt_epsilon;
    float half_sigma;
    int index; } iap;

  // load patch 1
  if ( blocki + threadIdx.x < myPatchPair.patch1_force_size ) {
    int i = myPatchPair.patch1_atom_start + blocki + threadIdx.x;
    float4 tmpa = ((float4*)atoms)[i];

    ipq.position.x = tmpa.x + myPatchPair.offset.x;
    ipq.position.y = tmpa.y + myPatchPair.offset.y;
    ipq.position.z = tmpa.z + myPatchPair.offset.z;
    ipq.charge = tmpa.w;

    uint4 tmpap = ((uint4*)atom_params)[i];

    iap.sqrt_epsilon = __int_as_float(tmpap.x);
    iap.half_sigma = __int_as_float(tmpap.y);
    iap.index = tmpap.z;
  }

  // avoid syncs by having all warps load pairlist
  if ( plcutoff2 == 0 ) {
    int i_pl = (blocki >> 2) + myPatchPair.block_flags_start;
    plu.i[threadIdx.x] = block_flags[i_pl + (threadIdx.x & 31)];
  } else {
    plu.i[threadIdx.x] = 0;
  }
  int pli = 4 * ( threadIdx.x & 96 );

  float4 ife, ife_slow;
  ife.x = 0.f;
  ife.y = 0.f;
  ife.z = 0.f;
  ife.w = 0.f;
  ife_slow.x = 0.f;
  ife_slow.y = 0.f;
  ife_slow.z = 0.f;
  ife_slow.w = 0.f;

  for ( int blockj = 0;
        blockj < myPatchPair.patch2_size;
        blockj += SHARED_SIZE, ++pli ) {

#ifdef __DEVICE_EMULATION__
  if ( plcutoff2 == 0 && threadIdx.x == 0 ) printf("%d %d %d %d %d %d %d\n", blockIdx.x, blocki, blockj, pli, pl[pli], (pli+128)&255, pl[(pli+128)&255]);
#endif
  if ( plcutoff2 == 0 && pl[pli] == 0 ) continue;

  int shared_size = myPatchPair.patch2_size - blockj;
  if ( shared_size > SHARED_SIZE ) shared_size = SHARED_SIZE;

  // load patch 2
  __syncthreads();

  if ( threadIdx.x < 4 * shared_size ) {
    int j = myPatchPair.patch2_atom_start + blockj;
    jpqu.i[threadIdx.x] = ((unsigned int *)(atoms + j))[threadIdx.x];
    japu.i[threadIdx.x] = ((unsigned int *)(atom_params + j))[threadIdx.x];
  }
  __syncthreads();

  if ( plcutoff2 == 0 && (pl[pli] & (1 << (threadIdx.x >> 5))) == 0 ) continue;

  // calc forces on patch 1
  if ( blocki + threadIdx.x < myPatchPair.patch1_force_size ) {

// be careful not to use // comments inside macros!
#define FORCE_INNER_LOOP(IPQ,IAP,DO_SLOW,DO_PAIRLIST) \
    for ( int j = 0; j < shared_size; ++j ) { \
      /* actually calculate force */ \
      float tmpx = jpqs[j].position.x - IPQ.position.x; \
      float tmpy = jpqs[j].position.y - IPQ.position.y; \
      float tmpz = jpqs[j].position.z - IPQ.position.z; \
      float r2 = tmpx*tmpx + tmpy*tmpy + tmpz*tmpz; \
      DO_PAIRLIST \
      if ( r2 < cutoff2 ) { \
        float4 fi = tex1D(force_table, rsqrtf(r2)); \
        bool excluded = false; \
        int indexdiff = (int)(IAP.index) - (int)(japs[j].index); \
        if ( abs(indexdiff) <= (int) japs[j].excl_maxdiff ) { \
          indexdiff += japs[j].excl_index; \
          excluded = ((exclusions[indexdiff>>5] & (1<<(indexdiff&31))) != 0); \
        } \
        float e = IAP.half_sigma + japs[j].half_sigma;  /* sigma */ \
        e *= e*e;  /* sigma^3 */ \
        e *= e;  /* sigma^6 */ \
        e *= ( e * fi.z + fi.y );  /* s^12 * fi.z - s^6 * fi.y */ \
        e *= IAP.sqrt_epsilon * japs[j].sqrt_epsilon;  /* full L-J */ \
        float e_slow = IPQ.charge * jpqs[j].charge; \
        e += e_slow * fi.x; \
        if ( DO_SLOW ) e_slow *= fi.w; \
        if ( ! excluded ) { \
          ife.w += r2 * e; \
          ife.x += tmpx * e; \
          ife.y += tmpy * e; \
          ife.z += tmpz * e; \
          if ( DO_SLOW ) { \
          ife_slow.w += r2 * e_slow; \
          ife_slow.x += tmpx * e_slow; \
          ife_slow.y += tmpy * e_slow; \
          ife_slow.z += tmpz * e_slow; \
          } \
        } \
      } }  /* cutoff */ \
    } /* end of FORCE_INNER_LOOP macro */

    if ( plcutoff2 == 0 ) {  // use pairlist
      if ( doSlow ) {
        FORCE_INNER_LOOP(ipq,iap,1,{)
      } else {
        FORCE_INNER_LOOP(ipq,iap,0,{)
      }
    } else {  // create pairlist
      bool plpli = 0;
      if ( doSlow ) {
        FORCE_INNER_LOOP(ipq,iap,1,if(r2<plcutoff2){plpli=1;)
      } else {
        FORCE_INNER_LOOP(ipq,iap,0,if(r2<plcutoff2){plpli=1;)
      }
      if ( plpli ) pl[pli] = 1;
    }

  } // if
  } // blockj loop

  if ( blocki + threadIdx.x < myPatchPair.patch1_force_size ) {
    int i_out = myPatchPair.patch1_force_start + blocki + threadIdx.x;
    force_buffers[i_out] = ife;
    if ( doSlow ) {
      slow_force_buffers[i_out] = ife_slow;
    }
    // accumulate net force to shared memory, warp-synchronous
    const int subwarp = threadIdx.x >> 2;  // 32 entries in table
    const int thread = threadIdx.x & 3;  // 4 threads share each entry
    for ( int g = 0; g < 4; ++g ) {
      if ( thread == g ) {
        sumf.a2d[subwarp][0] += ife.x;
        sumf.a2d[subwarp][1] += ife.y;
        sumf.a2d[subwarp][2] += ife.z;
        if ( doSlow ) {
          sumf_slow.a2d[subwarp][0] += ife_slow.x;
          sumf_slow.a2d[subwarp][1] += ife_slow.y;
          sumf_slow.a2d[subwarp][2] += ife_slow.z;
        }
      }
    }
  }
  if ( plcutoff2 != 0 ) {
    __syncthreads();  // all shared pairlist writes complete
    unsigned int pltmp;
    if ( threadIdx.x < 32 ) {
      pltmp = plu.i[threadIdx.x];
      pltmp |= plu.i[threadIdx.x+32] << 1;
      pltmp |= plu.i[threadIdx.x+64] << 2;
      pltmp |= plu.i[threadIdx.x+96] << 3;
    }
    __syncthreads();  // all shared pairlist reads complete
    if ( threadIdx.x < 32 ) {
      int i_pl = (blocki >> 2) + myPatchPair.block_flags_start;
      block_flags[i_pl + threadIdx.x] = pltmp;
    }
  }

  } // blocki loop

  __syncthreads();
  if ( threadIdx.x < 24 ) { // reduce forces, warp-synchronous
                            // 3 components, 8 threads per component
    const int i_out = myPatchPair.virial_start + threadIdx.x;
    {
      float f;
      f = sumf.a1d[threadIdx.x] + sumf.a1d[threadIdx.x + 24] + 
          sumf.a1d[threadIdx.x + 48] + sumf.a1d[threadIdx.x + 72];
      sumf.a1d[threadIdx.x] = f;
      f += sumf.a1d[threadIdx.x + 12];
      sumf.a1d[threadIdx.x] = f;
      f += sumf.a1d[threadIdx.x + 6];
      sumf.a1d[threadIdx.x] = f;
      f += sumf.a1d[threadIdx.x + 3];
      f *= 0.5f;  // compensate for double-counting
      // calculate virial contribution on first 3 threads
      sumf.a2d[threadIdx.x][0] = f * myPatchPair.offset.x;
      sumf.a2d[threadIdx.x][1] = f * myPatchPair.offset.y;
      sumf.a2d[threadIdx.x][2] = f * myPatchPair.offset.z;
      if ( threadIdx.x < 9 ) {  // write out output buffer
        virial_buffers[i_out] = sumf.a1d[threadIdx.x];
      }
    }
    if ( doSlow ) { // repeat above for slow forces
      float fs;
      fs = sumf_slow.a1d[threadIdx.x] + sumf_slow.a1d[threadIdx.x + 24] + 
           sumf_slow.a1d[threadIdx.x + 48] + sumf_slow.a1d[threadIdx.x + 72];
      sumf_slow.a1d[threadIdx.x] = fs;
      fs += sumf_slow.a1d[threadIdx.x + 12];
      sumf_slow.a1d[threadIdx.x] = fs;
      fs += sumf_slow.a1d[threadIdx.x + 6];
      sumf_slow.a1d[threadIdx.x] = fs;
      fs += sumf_slow.a1d[threadIdx.x + 3];
      fs *= 0.5f;
      sumf_slow.a2d[threadIdx.x][0] = fs * myPatchPair.offset.x;
      sumf_slow.a2d[threadIdx.x][1] = fs * myPatchPair.offset.y;
      sumf_slow.a2d[threadIdx.x][2] = fs * myPatchPair.offset.z;
      if ( threadIdx.x < 9 ) {
        slow_virial_buffers[i_out] = sumf_slow.a1d[threadIdx.x];
      }
    }
  }

 } // end of nonbonded calc

 { // start of force sum

  // make sure forces are visible in global memory
  __threadfence();

  __shared__ bool sumForces;

  if (threadIdx.x == 0) {
    int fli = myPatchPair.patch1_force_list_index;
    int fls = myPatchPair.patch1_force_list_size;
    int old = atomicInc(force_list_counters+fli,fls-1);
    sumForces = ( old == fls - 1 );
  }

  __syncthreads();

  if ( sumForces ) {
    dev_sum_forces(myPatchPair.patch1_force_list_index,
       atoms,force_lists,force_buffers,
       virial_buffers,forces,virials);

    if ( doSlow ) {
      dev_sum_forces(myPatchPair.patch1_force_list_index,
         atoms,force_lists,slow_force_buffers,
         slow_virial_buffers,slow_forces,slow_virials);
    }
  }

 } // end of force sum
}


__device__ __forceinline__ static void dev_sum_forces(
        const int force_list_index,
	const atom *atoms,
	const force_list *force_lists,
	const float4 *force_buffers,
	const float *virial_buffers,
	float4 *forces, float *virials) {
// call with one block per patch
// call BLOCK_SIZE threads per block
// call with no shared memory

  #define myForceList fl.fl
  __shared__ union {
    force_list fl;
    unsigned int i[FORCE_LIST_SIZE];
  } fl;

  if ( threadIdx.x < FORCE_LIST_USED ) {
    unsigned int tmp = ((unsigned int*)force_lists)[
                        FORCE_LIST_SIZE*force_list_index+threadIdx.x];
    fl.i[threadIdx.x] = tmp;
  }

  volatile __shared__ union {
    float a3d[32][3][3];
    float a2d[32][9];
    float a1d[32*9];
  } virial;

  for ( int i = threadIdx.x; i < 32*9; i += BLOCK_SIZE ) {
    virial.a1d[i] = 0.f;
  }

  __syncthreads();

  float vxx = 0.f;
  float vxy = 0.f;
  float vxz = 0.f;
  float vyx = 0.f;
  float vyy = 0.f;
  float vyz = 0.f;
  float vzx = 0.f;
  float vzy = 0.f;
  float vzz = 0.f;

  for ( int j = threadIdx.x; j < myForceList.patch_size; j += BLOCK_SIZE ) {

    const float4 *fbuf = force_buffers + myForceList.force_list_start + j;
    float4 fout;
    fout.x = 0.f;
    fout.y = 0.f;
    fout.z = 0.f;
    fout.w = 0.f;
    for ( int i=0; i < myForceList.force_list_size; ++i ) {
      float4 f = *fbuf;
      fout.x += f.x;
      fout.y += f.y;
      fout.z += f.z;
      fout.w += f.w;
      fbuf += myForceList.patch_stride;
    }

    // compiler will use st.global.f32 instead of st.global.v4.f32
    // if forcedest is directly substituted in the assignment
    const int forcedest = myForceList.force_output_start + j;
    forces[forcedest] = fout;

    float4 pos = ((float4*)atoms)[myForceList.atom_start + j];

    // accumulate per-atom virials to registers
    vxx += fout.x * pos.x;
    vxy += fout.x * pos.y;
    vxz += fout.x * pos.z;
    vyx += fout.y * pos.x;
    vyy += fout.y * pos.y;
    vyz += fout.y * pos.z;
    vzx += fout.z * pos.x;
    vzy += fout.z * pos.y;
    vzz += fout.z * pos.z;

  }

  { // accumulate per-atom virials to shared memory, warp-synchronous
    const int subwarp = threadIdx.x >> 2;  // 32 entries in table
    const int thread = threadIdx.x & 3;  // 4 threads share each entry
    for ( int g = 0; g < 4; ++g ) {
      if ( thread == g ) {
        virial.a3d[subwarp][0][0] += vxx;
        virial.a3d[subwarp][0][1] += vxy;
        virial.a3d[subwarp][0][2] += vxz;
        virial.a3d[subwarp][1][0] += vyx;
        virial.a3d[subwarp][1][1] += vyy;
        virial.a3d[subwarp][1][2] += vyz;
        virial.a3d[subwarp][2][0] += vzx;
        virial.a3d[subwarp][2][1] += vzy;
        virial.a3d[subwarp][2][2] += vzz;
      }
    }
  }
  __syncthreads();
  { // accumulate per-compute virials to shared memory, data-parallel
    const int halfwarp = threadIdx.x >> 4;  // 8 half-warps
    const int thread = threadIdx.x & 15;
    if ( thread < 9 ) {
      for ( int i = halfwarp; i < myForceList.force_list_size; i += 8 ) {
        virial.a2d[halfwarp][thread] +=
          virial_buffers[myForceList.virial_list_start + 16*i + thread];
      }
    }
  }
  __syncthreads();
  { // reduce virials in shared memory, warp-synchronous
    const int subwarp = threadIdx.x >> 3;  // 16 quarter-warps
    const int thread = threadIdx.x & 7;  // 8 threads per component
    if ( subwarp < 9 ) {  // 9 components
      float v;
      v = virial.a2d[thread][subwarp] + virial.a2d[thread+8][subwarp] +
          virial.a2d[thread+16][subwarp] + virial.a2d[thread+24][subwarp];
      virial.a2d[thread][subwarp] = v;
      v += virial.a2d[thread+4][subwarp];
      virial.a2d[thread][subwarp] = v;
      v += virial.a2d[thread+2][subwarp];
      virial.a2d[thread][subwarp] = v;
      v += virial.a2d[thread+1][subwarp];
      virial.a2d[thread][subwarp] = v;
    }
  }
  __syncthreads();
  if ( threadIdx.x < 9 ) {  // 9 components
    virials[myForceList.virial_output_start + threadIdx.x] =
                                              virial.a2d[0][threadIdx.x];
  }

}


void cuda_nonbonded_forces(float3 lata, float3 latb, float3 latc,
		float cutoff2, float plcutoff2,
		int cbegin, int ccount, int pbegin, int pcount,
		int doSlow, int usePairlists, int savePairlists) {

 if ( ccount ) {
   if ( usePairlists ) {
     if ( ! savePairlists ) plcutoff2 = 0.;
   } else {
     plcutoff2 = cutoff2;
   }
   int grid_dim = 65535;  // maximum allowed
   for ( int cstart = 0; cstart < ccount; cstart += grid_dim ) {
     if ( grid_dim > ccount - cstart ) grid_dim = ccount - cstart;
     // printf("%d %d %d\n",cbegin+cstart,grid_dim,patch_pairs_size);
     dev_nonbonded<<< grid_dim, BLOCK_SIZE, 0, stream
	>>>(patch_pairs+cbegin+cstart,atoms,atom_params,force_buffers,
	     (doSlow?slow_force_buffers:0), block_flags,
             virial_buffers, (doSlow?slow_virial_buffers:0),
             force_list_counters, force_lists,
             forces, virials,
             (doSlow?slow_forces:0), (doSlow?slow_virials:0),
	     lata, latb, latc, cutoff2, plcutoff2, doSlow);
     cuda_errcheck("dev_nonbonded");
   }
 }

#if 0
 if ( pcount ) {
  // printf("%d %d %d\n",pbegin,pcount,force_lists_size);
  dev_sum_forces<<< pcount, BLOCK_SIZE, 0, stream
	>>>(atoms,force_lists+pbegin,force_buffers,
                virial_buffers,forces,virials);
  if ( doSlow ) {
    dev_sum_forces<<< pcount, BLOCK_SIZE, 0, stream
	>>>(atoms,force_lists+pbegin,slow_force_buffers,
                slow_virial_buffers,slow_forces,slow_virials);
  }
  cuda_errcheck("dev_sum_forces");
 }
#endif

}


int cuda_stream_finished() {
  return ( hipStreamQuery(stream) == hipSuccess );
}


#endif  // NAMD_CUDA

