#include "hip/hip_runtime.h"

#include "ComputeNonbondedCUDAKernel.h"
#include <stdio.h>

#ifdef NAMD_CUDA


__constant__ unsigned int exclusions[MAX_EXCLUSIONS];

#define SET_EXCL(EXCL,BASE,DIFF) \
         (EXCL)[((BASE)+(DIFF))>>5] |= (1<<(((BASE)+(DIFF))&31))

void cuda_bind_exclusions(const unsigned int *t, int n) {

  hipMemcpyToSymbol(HIP_SYMBOL(exclusions), t, n*sizeof(unsigned int), 0);
  cuda_errcheck("memcpy to exclusions");
}


texture<float4, 1, hipReadModeElementType> force_table;

void cuda_bind_force_table(const float4 *t) {
    static hipArray *ct;
    if ( ! ct ) {
      hipMallocArray(&ct, &force_table.channelDesc, FORCE_TABLE_SIZE, 1);
      cuda_errcheck("allocating force table");
    }     hipMemcpyToArray(ct, 0, 0, t, FORCE_TABLE_SIZE*sizeof(float4), hipMemcpyHostToDevice);     // hipMemcpy(ct, t, FORCE_TABLE_SIZE*sizeof(float4), hipMemcpyHostToDevice);
    cuda_errcheck("memcpy to force table");

    force_table.normalized = true;
    force_table.addressMode[0] = hipAddressModeClamp;
    force_table.addressMode[1] = hipAddressModeClamp;
    force_table.filterMode = hipFilterModeLinear;

    hipBindTextureToArray(force_table, ct);
    cuda_errcheck("binding force table to texture");
}

static int patch_pairs_size;
static patch_pair *patch_pairs;

static int force_lists_size;
static force_list *force_lists;

static int force_buffers_size;
static float4 *force_buffers;
static float4 *slow_force_buffers;

static int atoms_size;
static atom *atoms;
static atom_param *atom_params;
static float4 *forces;
static float4 *slow_forces;

static int patch_pairs_alloc;
static int force_buffers_alloc;
static int force_lists_alloc;
static int atoms_alloc;

static int max_atoms_per_patch;

// static hipStream_t stream;
hipStream_t stream;
 
void cuda_init() {
  forces = 0;
  slow_forces = 0;
  atom_params = 0;
  atoms = 0;
  force_buffers = 0;
  slow_force_buffers = 0;
  force_lists = 0;
  patch_pairs = 0;

  patch_pairs_alloc = 0;
  force_buffers_alloc = 0;
  force_lists_alloc = 0;
  atoms_alloc = 0;

  hipStreamCreate(&stream);
  cuda_errcheck("hipStreamCreate");
}

void cuda_bind_patch_pairs(const patch_pair *pp, int npp,
                        const force_list *fl, int nfl,
                        int atoms_size_p, int force_buffers_size_p,
			int max_atoms_per_patch_p) {

  patch_pairs_size = npp;
  force_buffers_size = force_buffers_size_p;
  force_lists_size = nfl;
  atoms_size = atoms_size_p;
  max_atoms_per_patch = max_atoms_per_patch_p;

#if 0
 printf("%d %d %d %d %d %d %d %d\n",
      patch_pairs_size , patch_pairs_alloc ,
      force_buffers_size , force_buffers_alloc ,
      force_lists_size , force_lists_alloc ,
      atoms_size , atoms_alloc );
#endif

 if ( patch_pairs_size > patch_pairs_alloc ||
      force_buffers_size > force_buffers_alloc ||
      force_lists_size > force_lists_alloc ||
      atoms_size > atoms_alloc ) {

  patch_pairs_alloc = (int) (1.2 * patch_pairs_size);
  force_buffers_alloc = (int) (1.2 * force_buffers_size);
  force_lists_alloc = (int) (1.2 * force_lists_size);
  atoms_alloc = (int) (1.2 * atoms_size);

  if ( forces ) hipFree(forces);
  if ( slow_forces ) hipFree(slow_forces);
  if ( atom_params ) hipFree(atom_params);
  if ( atoms ) hipFree(atoms);
  if ( force_buffers ) hipFree(force_buffers);
  if ( slow_force_buffers ) hipFree(slow_force_buffers);
  if ( force_lists ) hipFree(force_lists);
  if ( patch_pairs ) hipFree(patch_pairs);
  cuda_errcheck("free everything");

#if 1
  int totalmem = patch_pairs_alloc * sizeof(patch_pair) +
		force_lists_alloc * sizeof(force_list) +
		2 * force_buffers_alloc * sizeof(float4) +
		atoms_alloc * sizeof(atom) +
		atoms_alloc * sizeof(atom_param) +
		2 * atoms_alloc * sizeof(float4);
  // printf("allocating %d MB of memory on GPU\n", totalmem >> 20);
#endif

  hipMalloc((void**) &patch_pairs, patch_pairs_alloc * sizeof(patch_pair));
  hipMalloc((void**) &force_lists, force_lists_alloc * sizeof(force_list));
  hipMalloc((void**) &force_buffers, force_buffers_alloc * sizeof(float4));
  hipMalloc((void**) &slow_force_buffers, force_buffers_alloc * sizeof(float4));
  hipMalloc((void**) &atoms, atoms_alloc * sizeof(atom));
  hipMalloc((void**) &atom_params, atoms_alloc * sizeof(atom_param));
  hipMalloc((void**) &forces, atoms_alloc * sizeof(float4));
  hipMalloc((void**) &slow_forces, atoms_alloc * sizeof(float4));
  cuda_errcheck("malloc everything");

 }

  hipMemcpy(patch_pairs, pp, npp * sizeof(patch_pair),
				hipMemcpyHostToDevice);
  cuda_errcheck("memcpy to patch_pairs");

  hipMemcpy(force_lists, fl, nfl * sizeof(force_list),
				hipMemcpyHostToDevice);
  cuda_errcheck("memcpy to force_lists");
}

void cuda_bind_atom_params(const atom_param *t) {
  hipMemcpyAsync(atom_params, t, atoms_size * sizeof(atom_param),
				hipMemcpyHostToDevice, stream);
  cuda_errcheck("memcpy to atom_params");
}

void cuda_bind_atoms(const atom *a) {
  cuda_errcheck("before memcpy to atoms");
  hipMemcpyAsync(atoms, a, atoms_size * sizeof(atom),
				hipMemcpyHostToDevice, stream);
  cuda_errcheck("memcpy to atoms");
}

void cuda_load_forces(float4 *f, float4 *f_slow, int begin, int count) {
  // printf("load forces %d %d %d\n",begin,count,atoms_size);
  hipMemcpyAsync(f+begin, forces+begin, count * sizeof(float4),
				hipMemcpyDeviceToHost, stream);
  if ( f_slow ) {
    hipMemcpyAsync(f_slow+begin, slow_forces+begin, count * sizeof(float4),
				hipMemcpyDeviceToHost, stream);
  }
  cuda_errcheck("memcpy from forces");
}


#if 0
__host__ __device__ static int3 patch_coords_from_id(
        dim3 PATCH_GRID, int id) {

  return make_int3( id % PATCH_GRID.x,
                ( id / PATCH_GRID.x ) % PATCH_GRID.y,
                id / ( PATCH_GRID.x * PATCH_GRID.y ) );
}

__host__ __device__ static int patch_id_from_coords(
        dim3 PATCH_GRID, int3 coords) {

  // handles periodic boundaries
  int x = (coords.x + 4 * PATCH_GRID.x) % PATCH_GRID.x;
  int y = (coords.y + 4 * PATCH_GRID.y) % PATCH_GRID.y;
  int z = (coords.z + 4 * PATCH_GRID.z) % PATCH_GRID.z;

  return ( z * PATCH_GRID.y + y ) * PATCH_GRID.x + x;
}

__host__ __device__ static int3 patch_offset_from_neighbor(int neighbor) {

  // int3 coords = patch_coords_from_id(make_uint3(3,3,3), 13 + neighbor);
  int3 coords = patch_coords_from_id(make_uint3(3,3,3), neighbor);
  return make_int3(coords.x - 1, coords.y - 1, coords.z - 1);

}
#endif
 
#define BLOCK_SIZE 128
#define SHARED_SIZE 32

__global__ static void dev_nonbonded(
	const patch_pair *patch_pairs,
	const atom *atoms,
	const atom_param *atom_params,
	float4 *force_buffers,
	float4 *slow_force_buffers,
        float3 lata, float3 latb, float3 latc,
	float cutoff2) {
// call with two blocks per patch_pair
// call with BLOCK_SIZE threads per block
// call with no shared memory

  #define jpqs jpqu.d
  __shared__ union {
    atom d[SHARED_SIZE];
    unsigned int i[BLOCK_SIZE];
    float f[BLOCK_SIZE];
  } jpqu;

  #define japs japu.d
  __shared__ union {
    atom_param d[SHARED_SIZE];
    unsigned int i[BLOCK_SIZE];
  } japu;

  #define myPatchPair pp.pp
  __shared__ union { patch_pair pp; unsigned int i[PATCH_PAIR_SIZE]; } pp;

  if ( threadIdx.x < PATCH_PAIR_USED ) {
    unsigned int tmp = ((unsigned int*)patch_pairs)[
			(sizeof(patch_pair)>>2)*blockIdx.x+threadIdx.x];
    pp.i[threadIdx.x] = tmp;
  }
  __syncthreads();

  // convert scaled offset with current lattice
  if ( threadIdx.x == 0 ) {
    float offx = myPatchPair.offset.x * lata.x
               + myPatchPair.offset.y * latb.x
               + myPatchPair.offset.z * latc.x;
    float offy = myPatchPair.offset.x * lata.y
               + myPatchPair.offset.y * latb.y
               + myPatchPair.offset.z * latc.y;
    float offz = myPatchPair.offset.x * lata.z
               + myPatchPair.offset.y * latb.z
               + myPatchPair.offset.z * latc.z;
    myPatchPair.offset.x = offx;
    myPatchPair.offset.y = offy;
    myPatchPair.offset.z = offz;
  }

  __syncthreads();

  for ( int blocki = 0;
        blocki < myPatchPair.patch1_force_size;
        blocki += BLOCK_SIZE ) {

  atom ipq;
  struct {
    float sqrt_epsilon;
    float half_sigma;
    int index; } iap;

  // load patch 1
  if ( blocki + threadIdx.x < myPatchPair.patch1_force_size ) {
    int i = myPatchPair.patch1_atom_start + blocki + threadIdx.x;
    float4 tmpa = ((float4*)atoms)[i];

    ipq.position.x = tmpa.x + myPatchPair.offset.x;
    ipq.position.y = tmpa.y + myPatchPair.offset.y;
    ipq.position.z = tmpa.z + myPatchPair.offset.z;
    ipq.charge = tmpa.w;

    uint4 tmpap = ((uint4*)atom_params)[i];

    iap.sqrt_epsilon = __int_as_float(tmpap.x);
    iap.half_sigma = __int_as_float(tmpap.y);
    iap.index = tmpap.z;
  }

  float4 ife, ife_slow;
  ife.x = 0.f;
  ife.y = 0.f;
  ife.z = 0.f;
  ife.w = 0.f;
  ife_slow.x = 0.f;
  ife_slow.y = 0.f;
  ife_slow.z = 0.f;
  ife_slow.w = 0.f;

  for ( int blockj = 0;
        blockj < myPatchPair.patch2_size;
        blockj += SHARED_SIZE ) {

  int shared_size = myPatchPair.patch2_size - blockj;
  if ( shared_size > SHARED_SIZE ) shared_size = SHARED_SIZE;

  // load patch 2
  // sync needed because of loop, could avoid with double-buffering
  __syncthreads();

  if ( threadIdx.x < 4 * shared_size ) {
    int j = myPatchPair.patch2_atom_start + blockj;
    jpqu.i[threadIdx.x] = ((unsigned int *)(atoms + j))[threadIdx.x];
    japu.i[threadIdx.x] = ((unsigned int *)(atom_params + j))[threadIdx.x];
  }
  __syncthreads();

  // calc forces on patch 1
  if ( blocki + threadIdx.x < myPatchPair.patch1_force_size ) {

// be careful not to use // comments inside macros!
#define FORCE_INNER_LOOP(IPQ,IAP,DO_SLOW) \
    for ( int j = 0; j < shared_size; ++j ) { \
      /* actually calculate force */ \
      float tmpx = jpqs[j].position.x - IPQ.position.x; \
      float tmpy = jpqs[j].position.y - IPQ.position.y; \
      float tmpz = jpqs[j].position.z - IPQ.position.z; \
      float r2 = tmpx*tmpx + tmpy*tmpy + tmpz*tmpz; \
      if ( r2 < cutoff2 ) { \
        float4 fi = tex1D(force_table, rsqrtf(r2)); \
        bool excluded = false; \
        int indexdiff = (int)(IAP.index) - (int)(japs[j].index); \
        if ( abs(indexdiff) <= (int) japs[j].excl_maxdiff ) { \
          indexdiff += japs[j].excl_index; \
          excluded = ((exclusions[indexdiff>>5] & (1<<(indexdiff&31))) != 0); \
        } \
        float e = IAP.half_sigma + japs[j].half_sigma;  /* sigma */ \
        e *= e*e;  /* sigma^3 */ \
        e *= e;  /* sigma^6 */ \
        e *= ( e * fi.z + fi.y );  /* s^12 * fi.z - s^6 * fi.y */ \
        e *= IAP.sqrt_epsilon * japs[j].sqrt_epsilon;  /* full L-J */ \
        float e_slow = IPQ.charge * jpqs[j].charge; \
        e += e_slow * fi.x; \
        if ( DO_SLOW ) e_slow *= fi.w; \
        if ( ! excluded ) { \
          ife.w += r2 * e; \
          ife.x += tmpx * e; \
          ife.y += tmpy * e; \
          ife.z += tmpz * e; \
          if ( DO_SLOW ) { \
          ife_slow.w += r2 * e_slow; \
          ife_slow.x += tmpx * e_slow; \
          ife_slow.y += tmpy * e_slow; \
          ife_slow.z += tmpz * e_slow; \
          } \
        } \
      }  /* cutoff */ \
    } /* end of FORCE_INNER_LOOP macro */

    if ( slow_force_buffers ) {
      FORCE_INNER_LOOP(ipq,iap,1)
    } else {
      FORCE_INNER_LOOP(ipq,iap,0)
    }

  } // if
  } // blockj loop

  if ( blocki + threadIdx.x < myPatchPair.patch1_force_size ) {
    int i_out = myPatchPair.patch1_force_start + blocki + threadIdx.x;
    force_buffers[i_out] = ife;
    if ( slow_force_buffers ) {
      slow_force_buffers[i_out] = ife_slow;
    }
  }

  } // blocki loop

}


__global__ static void dev_sum_forces(
	const force_list *force_lists,
	const float4 *force_buffers,
	float4 *forces) {
// call with one block per patch
// call multiple of 64 threads per block
// call with no shared memory

  __shared__ force_list myForceList;

  if ( threadIdx.x == 0 ) {
    myForceList = force_lists[blockIdx.x];
  }
  __syncthreads();

  for ( int j = threadIdx.x; j < myForceList.patch_size; j += blockDim.x ) {

    const float4 *fbuf = force_buffers + myForceList.force_list_start + j;
    float4 fout;
    fout.x = 0.f;
    fout.y = 0.f;
    fout.z = 0.f;
    fout.w = 0.f;
    for ( int i=0; i < myForceList.force_list_size; ++i ) {
      float4 f = *fbuf;
      fout.x += f.x;
      fout.y += f.y;
      fout.z += f.z;
      fout.w += f.w;
      fbuf += myForceList.patch_size;
    }

    forces[myForceList.force_output_start + j] = fout;

  }
}


void cuda_nonbonded_forces(float3 lata, float3 latb, float3 latc, float cutoff2,
		int cbegin, int ccount, int pbegin, int pcount, int doSlow) {

 if ( ccount ) {
   int grid_dim = 65535;  // maximum allowed
   for ( int cstart = 0; cstart < ccount; cstart += grid_dim ) {
     if ( grid_dim > ccount - cstart ) grid_dim = ccount - cstart;
     // printf("%d %d %d\n",cbegin+cstart,grid_dim,patch_pairs_size);
     dev_nonbonded<<< grid_dim, BLOCK_SIZE, 0, stream
	>>>(patch_pairs+cbegin+cstart,atoms,atom_params,force_buffers,
	     (doSlow?slow_force_buffers:0), lata, latb, latc, cutoff2);
     cuda_errcheck("dev_nonbonded");
   }
 }

 if ( pcount ) {
  // printf("%d %d %d\n",pbegin,pcount,force_lists_size);
  dev_sum_forces<<< pcount, 128, 0, stream
	>>>(force_lists+pbegin,force_buffers,forces);
  if ( doSlow ) {
    dev_sum_forces<<< pcount, 128, 0, stream
	>>>(force_lists+pbegin,slow_force_buffers,slow_forces);
  }
  cuda_errcheck("dev_sum_forces");
 }

}


int cuda_stream_finished() {
  return ( hipStreamQuery(stream) == hipSuccess );
}


#endif  // NAMD_CUDA

